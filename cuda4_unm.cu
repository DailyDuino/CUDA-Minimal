#include "hip/hip_runtime.h"

#include <stdio.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>

#include "cuda_img.h"





__global__ void copyImage_kernal( CudaImg sourceImage, CudaImg returnImage )
{
    // X,Y coordinates and check image dimensions
    //--- this remains the same ---//
    int l_y = blockDim.y * blockIdx.y + threadIdx.y;
    int l_x = blockDim.x * blockIdx.x + threadIdx.x;
    if ( l_y >= t_color_cuda_img.m_size.y ) return;
    if ( l_x >= t_color_cuda_img.m_size.x ) return;
    //--- this remains the same ---//

    //----- Your Code here -----////


    uchar3 pixel = sourceImage.getpixelRGB(l_y,l_x);
    returnImage.getpixelRGB(l_y,l_x) = pixel;


}

void copyImage( CudaImg sourceImage, CudaImg returnImage )
{
    hipError_t l_cerr;

    // Grid creation, size of grid must be equal or greater than images
    int l_block_size = 16;
    
    dim3 l_blocks( ( sourceImage.m_size.x + l_block_size - 1 ) / l_block_size, ( sourceImage.m_size.y + l_block_size - 1 ) / l_block_size );
    dim3 l_threads( l_block_size, l_block_size );


    // Calling kernal function below
    copyImage_kernal<<< l_blocks, l_threads >>>( sourceImage, returnImage ); 

    

    if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

    hipDeviceSynchronize();
}






